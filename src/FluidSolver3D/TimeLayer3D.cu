#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2011 Nikolai Sakharnykh
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

// GPU implementation of TimeLayer3D

#include "Grid3D.h"

#define COPY_BLOCK_DIM_X		32
#define COPY_BLOCK_DIM_Y		8

#define TRANSPOSE_SMEM_TILE_DIM			16
#define TRANSPOSE_SMEM_BLOCK_ROWS		16

#define TRANSPOSE_CACHE_TILE_DIM		32
#define TRANSPOSE_CACHE_BLOCK_ROWS		8

using namespace FluidSolver3D;

__global__ void copy_grid(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = nodes[id].v.x;
			v[id] = nodes[id].v.y;
			w[id] = nodes[id].v.z;
			T[id] = nodes[id].T;
		}
	}
}

__global__ void clear(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = const_u;
			v[id] = const_v;
			w[id] = const_w;
			T[id] = const_T;
		}
	}
}

__global__ void copy(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = src[id];
	}
}

__global__ void merge(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = ( dest[id] + src[id] ) / 2;
	}
}

__global__ void transpose_shared(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest)
{
	__shared__ FTYPE tile[TRANSPOSE_SMEM_TILE_DIM][TRANSPOSE_SMEM_TILE_DIM+1];

    // read the tile from global memory into shared memory
	int k0 = blockIdx.x * TRANSPOSE_SMEM_TILE_DIM + threadIdx.x;
	int j0 = blockIdx.y * TRANSPOSE_SMEM_TILE_DIM + threadIdx.y;

	int j1 = blockIdx.y * TRANSPOSE_SMEM_TILE_DIM + threadIdx.x;
    int k1 = blockIdx.x * TRANSPOSE_SMEM_TILE_DIM + threadIdx.y;

	int base_u0 = k0 + j0 * dimz;	
    int base_u1 = j1 + k1 * dimy;
	
	for (int i = 0; i < dimx; i++)
	{	
		// read tile from global to shared memory
		for (int row = 0; row < TRANSPOSE_SMEM_TILE_DIM; row += TRANSPOSE_SMEM_BLOCK_ROWS)
			tile[threadIdx.y + row][threadIdx.x] = src[base_u0 + row * dimz];
		base_u0 += dimz * dimy;
	
		__syncthreads();
	
		// write the transposed tile to global memory 
		for (int row = 0; row < TRANSPOSE_SMEM_TILE_DIM; row += TRANSPOSE_SMEM_BLOCK_ROWS)
			dest[base_u1 + row * dimy] = tile[threadIdx.x][threadIdx.y + row];
		base_u1 += dimz * dimy;
	}
}

__global__ void transpose_cache(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest)
{
	int i = blockIdx.y / ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM);
	int blkY = blockIdx.y % ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM);

	int j1 = blkY * TRANSPOSE_CACHE_TILE_DIM + threadIdx.x;
    int k1 = blockIdx.x * TRANSPOSE_CACHE_TILE_DIM + threadIdx.y;

	int base_src = blockIdx.x * TRANSPOSE_CACHE_TILE_DIM + blkY * TRANSPOSE_CACHE_TILE_DIM * dimz;
	int base_dst = j1 + k1 * dimy;
	
	base_src += dimz * dimy * i;
	base_dst += dimz * dimy * i;

	// load directly from global memory filling up L1
	for (int row = 0; row < TRANSPOSE_CACHE_TILE_DIM; row += TRANSPOSE_CACHE_BLOCK_ROWS)
		dest[base_dst + row * dimy] = src[base_src + threadIdx.x * dimz + threadIdx.y + row];
}

void CopyFromGrid_GPU(int dimx, int dimy, int dimz, FTYPE **u, FTYPE **v, FTYPE **w, FTYPE **T, Node **nodes, NodeType target, int haloSize = 0)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		copy_grid<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, u[i] + haloSize, v[i] + haloSize, w[i] + haloSize, T[i] + haloSize, nodes[i], target);
	}
	hipDeviceSynchronize();
}

void CopyFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE **src, FTYPE **dest, Node **nodes, NodeType target, int haloSize = 0)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		copy<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, src[i] + haloSize, dest[i] + haloSize, nodes[i], target);
	}
	hipDeviceSynchronize();
}

void MergeFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE **src, FTYPE **dest, Node **nodes, NodeType target,  int haloSize = 0)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		merge<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, src[i] + haloSize, dest[i] + haloSize, nodes[i], target);
	}
	hipDeviceSynchronize();
}

void Clear_GPU(int dimx, int dimy, int dimz, FTYPE **u, FTYPE **v, FTYPE **w, FTYPE **T, Node **nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T, int haloSize = 0)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		clear<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, u[i] + haloSize, v[i] + haloSize, w[i] + haloSize, T[i] + haloSize, nodes[i], target, const_u, const_v, const_w, const_T);
	}
	hipDeviceSynchronize();
}

void Transpose_GPU_shared(int dimx, int dimy, int dimz, FTYPE **u, FTYPE **dest_u, int haloSize = 0)
{
	dim3 block(TRANSPOSE_SMEM_TILE_DIM, TRANSPOSE_SMEM_BLOCK_ROWS);
	dim3 grid((dimz + TRANSPOSE_SMEM_TILE_DIM - 1)/TRANSPOSE_SMEM_TILE_DIM, (dimy + TRANSPOSE_SMEM_TILE_DIM - 1)/TRANSPOSE_SMEM_TILE_DIM);
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(transpose_shared), hipFuncCachePreferL1);
		transpose_shared<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, u[i] + haloSize, dest_u[i] + haloSize);
	}
	hipDeviceSynchronize();
}

void Transpose_GPU_cache(int dimx, int dimy, int dimz, FTYPE **u, FTYPE **dest_u, int haloSize = 0)
{
	dim3 block(TRANSPOSE_CACHE_TILE_DIM, TRANSPOSE_CACHE_BLOCK_ROWS);	
	GPUplan *pGPUplan = GPUplan::Instance();
	for (int i = 0; i < pGPUplan->size(); i++)
	{	
		hipSetDevice(i);
		dim3 grid((dimz + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM, pGPUplan->node(i)->getLength1D() * ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM));
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(transpose_cache), hipFuncCachePreferL1);
		transpose_cache<<<grid, block>>>(pGPUplan->node(i)->getLength1D(), dimy, dimz, u[i] + haloSize, dest_u[i] + haloSize);
	}
	hipDeviceSynchronize();
}

void Transpose_GPU(int dimx, int dimy, int dimz, FTYPE **u, FTYPE **dest_u, int haloSize = 0)
{
#if 1
	Transpose_GPU_shared(dimx, dimy, dimz, u, dest_u, haloSize);
#else
	Transpose_GPU_cache(dimx, dimy, dimz, u, dest_u, haloSize);
#endif
}
