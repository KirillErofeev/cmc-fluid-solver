#include "hip/hip_runtime.h"
// GPU implementation of TimeLayer3D

#include "Grid3D.h"

#define BLOCK_DIM_X		32
#define BLOCK_DIM_Y		8

#define TRANSPOSE_BLOCK_DIM		16

using namespace FluidSolver3D;

__global__ void copy_grid(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = nodes[id].v.x;
			v[id] = nodes[id].v.y;
			w[id] = nodes[id].v.z;
			T[id] = nodes[id].T;
		}
	}
}

__global__ void clear(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = const_u;
			v[id] = const_v;
			w[id] = const_w;
			T[id] = const_T;
		}
	}
}

__global__ void copy(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = src[id];
	}
}

__global__ void merge(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = ( dest[id] + src[id] ) / 2;
	}
}

__global__ void transpose(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest)
{
	__shared__ FTYPE block[TRANSPOSE_BLOCK_DIM][TRANSPOSE_BLOCK_DIM+1];

    // read the tile from global memory into shared memory
	int k0 = blockIdx.x * TRANSPOSE_BLOCK_DIM + threadIdx.x;
	int j0 = blockIdx.y * TRANSPOSE_BLOCK_DIM + threadIdx.y;

	int j1 = blockIdx.y * TRANSPOSE_BLOCK_DIM + threadIdx.x;
    int k1 = blockIdx.x * TRANSPOSE_BLOCK_DIM + threadIdx.y;

	if (k0 >= dimz) k0 = dimz-1;
	if (j0 >= dimy) j0 = dimy-1;
	int base_u0 = k0 + j0 * dimz;	

	if (k1 >= dimz) k1 = dimz-1;
	if (j1 >= dimy) j1 = dimy-1;
    int base_u1 = j1 + k1 * dimy;
	
	for (int i = 0; i < dimx; i++)
	{	
		// read tile from global to shared memory
		block[threadIdx.y][threadIdx.x] = src[base_u0];
		base_u0 += dimz * dimy;
	
		__syncthreads();
	
		// write the transposed tile to global memory 
		dest[base_u1] = block[threadIdx.x][threadIdx.y];
		base_u1 += dimz * dimy;
	}
}


void CopyFromGrid_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target)
{
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	copy_grid<<<grid, block>>>(dimx, dimy, dimz, u, v, w, T, nodes, target);
	hipDeviceSynchronize();
}

void CopyFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	copy<<<grid, block>>>(dimx, dimy, dimz, src, dest, nodes, target);
	hipDeviceSynchronize();
}

void MergeFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	merge<<<grid, block>>>(dimx, dimy, dimz, src, dest, nodes, target);
	hipDeviceSynchronize();
}

void Clear_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T)
{
	dim3 block(BLOCK_DIM_X, BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	clear<<<grid, block>>>(dimx, dimy, dimz, u, v, w, T, nodes, target, const_u, const_v, const_w, const_T);
	hipDeviceSynchronize();
}

void Transpose_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *dest_u)
{
	dim3 block(TRANSPOSE_BLOCK_DIM, TRANSPOSE_BLOCK_DIM);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	transpose<<<grid, block>>>(dimx, dimy, dimz, u, dest_u);
	hipDeviceSynchronize();
}