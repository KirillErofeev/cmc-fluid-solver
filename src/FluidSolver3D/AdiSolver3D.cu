#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2011 Nikolai Sakharnykh
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "AdiSolver3D.h"

#define SEG_BLOCK_DIM_X		32
#define SEG_BLOCK_DIM_Y		8

#if( __CUDA_ARCH__ < 120 )
#define SOLVER_BLOCK_DIM	128
#else
#define SOLVER_BLOCK_DIM	256
#endif

namespace FluidSolver3D
{
	struct FluidParamsGPU
	{
		FTYPE vis_dx2;
		FTYPE dt, dx, dy, dz;
		FTYPE v_T, t_phi;

		FluidParamsGPU( VarType var, DirType dir, FTYPE _dt, FTYPE _dx, FTYPE _dy, FTYPE _dz, FluidParams _params ) : 
			dt(_dt), dx(_dx), dy(_dy), dz(_dz)
		{
			switch (var)
			{
			case type_U:
			case type_V:
			case type_W:	
				switch (dir)
				{
				case X:	vis_dx2 = _params.v_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.v_vis / (dy * dy); break;
				case Z: case Z_as_Y: vis_dx2 = _params.v_vis / (dz * dz); break;
				}
				break;
			case type_T:
				switch (dir)
				{
				case X:	vis_dx2 = _params.t_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.t_vis / (dy * dy); break;
				case Z: case Z_as_Y:  vis_dx2 = _params.t_vis / (dz * dz); break;
				}
				break;
			}
						
			v_T = _params.v_T;
			t_phi = _params.t_phi;
		}
	};

#if 1
	// interleave matrix arrays for better memory access
	#define get(a, elem_id)			a[id + (elem_id + 1) * max_n_max_n * MAX_SEGS_PER_ROW]
#else
	// sequential layout - bad access pattern,  currently not implemented for MGPU
//	#define get(a, elem_id)			a[elem_id + id * max_n]
#endif

	template<int dir, int var>
	__device__ void apply_bc0(int i, int j, int k, int dimy, int dimz, FTYPE &b0, FTYPE &c0, FTYPE &d0, Node &node, SegmentType segType = BOUND)
	{
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				return;
			case BOUND_END:
				return;
			};
		}

		//int id = i * dimy * dimz;
		//if (dir != Z_as_Y) id += j * dimz + k;
		//	else id += j * dimy + k;

		if ((var == type_T && node.bc_temp == BC_FREE) ||
			(var != type_T && node.bc_vel == BC_FREE))
		{
			// free: f(0) = 2 * f(1) - f(2)
			b0 = 2.0; 
			c0 = -1.0; 
			d0 = 0.0;
		}
		else
		{
			// no-slip: f(0) = f(1)
			b0 = 1.0; 
			c0 = 0.0; 

			switch (var)
			{
				case type_U: d0 = node.v.x; break;  
				case type_V: d0 = node.v.y; break;
				case type_W: d0 = node.v.z; break;
				case type_T: d0 = node.T; break;
			}
		}
	}

	template<int dir, int var>
	__device__ void apply_bc1(int i, int j, int k, int dimy, int dimz, FTYPE &a1, FTYPE &b1, FTYPE &d1, Node &node, SegmentType segType = BOUND)
	{
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				return;
			case BOUND_START:
				return;
			}
		}

		//int id = i * dimy * dimz;
		//if (dir != Z_as_Y) id += j * dimz + k;
		//	else id += j * dimy + k;

		if ((var == type_T && node.bc_temp == BC_FREE) ||
			(var != type_T && node.bc_vel == BC_FREE))
		{
			// free: f(N) = 2 * f(N-1) - f(N-2)
			a1 = -1.0;
			b1 = 2.0;
			d1 = 0.0;
		}
		else
		{
			// no-slip: f(N) = f(N-1)
			a1 = 0.0; 
			b1 = 1.0; 

			switch (var)
			{
				case type_U: d1 = node.v.x; break;
				case type_V: d1 = node.v.y; break;
				case type_W: d1 = node.v.z; break;
				case type_T: d1 = node.T; break;
			}
		}
	}

	template<int dir, int var>
	__device__ void build_matrix(FluidParamsGPU params, int i, int j, int k, FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, int n, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, int id, int num_seg, int max_n_max_n, SegmentType segType = BOUND)
	{	
		int start = 1;
		int end = n-1;
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				start = 0; end = n;				
				break;
			case BOUND_START:
				end = n;
				break;
			case BOUND_END:
				start = 0;
				break;
			}
		}

		for (int p = start; p < end; p++)
		{
			switch (dir)
			{
			case X:		
				get(a,p) = - temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i+p, j, k) * 3 / params.dt - params.v_T * temp.d_x(temp.T, params.dx, i+p, j, k); break;
				case type_V: get(d,p) = cur.elem(cur.v, i+p, j, k) * 3 / params.dt; break;
				case type_W: get(d,p) = cur.elem(cur.w, i+p, j, k) * 3 / params.dt; break;
				case type_T: get(d,p) = cur.elem(cur.T, i+p, j, k) * 3 / params.dt + params.t_phi * temp.DissFuncX(params.dx, params.dy, params.dz, i+p, j, k); break;
				}	
				break;

			case Y:
				get(a,p) = - temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, j+p, k) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, j+p, k) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dy, i, j+p, k); break;
				case type_W: get(d,p) = cur.elem(cur.w, i, j+p, k) * 3 / params.dt; break;
				case type_T: get(d,p) = cur.elem(cur.T, i, j+p, k) * 3 / params.dt + params.t_phi * temp.DissFuncY(params.dx, params.dy, params.dz, i, j+p, k); break;
				}
				break;

			case Z:
				get(a,p) = - temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, j, k+p) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, j, k+p) * 3 / params.dt; break;
				case type_W: get(d,p) = cur.elem(cur.w, i, j, k+p) * 3 / params.dt - params.v_T * temp.d_z(temp.T, params.dz, i, j, k+p); break;
				case type_T: get(d,p) = cur.elem(cur.T, i, j, k+p) * 3 / params.dt + params.t_phi * temp.DissFuncZ(params.dx, params.dy, params.dz, i, j, k+p); break;
				}
				break;

			case Z_as_Y:
				get(a,p) = - temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, k+p, j) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, k+p, j) * 3 / params.dt; break;
				case type_W: get(d,p) = cur.elem(cur.w, i, k+p, j) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dz, i, k+p, j); break;
				case type_T: get(d,p) = cur.elem(cur.T, i, k+p, j) * 3 / params.dt + params.t_phi * temp.DissFuncZ_as_Y(params.dx, params.dz, params.dy, i, k+p, j); break;
				}
				break;
			}
		}
	}

template<int dir, int swipe>
	__device__ void solve_tridiagonal(FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int num, int id, int num_seg, int max_n_max_n, int dimX, SegmentType segType = BOUND)
	{
		switch (swipe)
		{
			case ALL:
			case FORWARD:
			{
				int start = 1;

				switch (dir)
				{
				case X:
						switch (segType)
						{
						case UNBOUND:
							start = 0;
							break;
						case BOUND_END:
							start = 0;
							get(c, num-1) = 0.0;
							break;
						case BOUND:
							get(c, num-1) = 0.0;
							get(c,0) = get(c,0) / get(b,0);
							get(d,0) = get(d,0) / get(b,0);
							break;
						case BOUND_START:
							get(c,0) = get(c,0) / get(b,0);
							get(d,0) = get(d,0) / get(b,0);
							break;
						}
						break;
				default:
					get(c,0) = get(c,0) / get(b,0);
					get(d,0) = get(d,0) / get(b,0);
					get(c,num-1) = 0.0;
					break;
				}

				for (int i = start; i < num; i++)
				{
					get(c,i) = get(c,i) / (get(b,i) - get(a,i) * get(c,i-1));
					get(d,i) = (get(d,i) - get(d,i-1) * get(a,i)) / (get(b,i) - get(a,i) * get(c,i-1));
				}
				switch (dir)
				{
				case X:
					get(c, dimX-1) = get(c, num-1);
					get(d, dimX-1) = get(d, num-1);
					break;
				}
				break;
			}			
		}

		switch (swipe)
		{
			case ALL:
			case BACK:
			{
				int end = num - 1;

				switch (dir)
				{
				case X:
					switch (segType)
					{
					case UNBOUND:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_START:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_END:
						get(x, num-1) = get(d,num-1);
						break;
					case BOUND:
						get(x, num-1) = get(d,num-1);
						break;
					}
					break;
				default:
					get(x, num-1) = get(d,num-1);
					break;
				}

				for (int i = end-1; i >= 0; i--) 
					get(x,i) = get(d,i) -  get(c,i) * get(x, i+1);
				break;
			}
		}
	}

	template<int dir, int var>
	__device__ void update_segment( FTYPE *x, Segment3D &seg, NodeType *nodes, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &layer, int id, int num_seg, int max_n_max_n )
	{
		int i = seg.posx;
		int j = seg.posy;
		int k = seg.posz;

		if( dir == Z_as_Y ) 
		{
			k = seg.posy;
			j = seg.posz;
		}

		for (int t = 0; t < seg.size; t++)
		{
			switch (var)
			{
			case type_U: layer.elem(layer.u, i, j, k) = get(x,t); break;
			case type_V: layer.elem(layer.v, i, j, k) = get(x,t); break;
			case type_W: layer.elem(layer.w, i, j, k) = get(x,t); break;
			case type_T: layer.elem(layer.T, i, j, k) = get(x,t); break;
			}
#if INTERNAL_MERGE_ENABLE == 1
			//if (t == 0)
			//	if (nodes.first.type != NODE_IN)
			//		continue;
			//if (t == seg.size-1)
			//	if (nodes.last.type != NODE_IN)
			//		continue;

			int idn;
			idn = i * layer.dimy * layer.dimz;
			switch(dir)
			{
			case Z_as_Y:
				idn += k * layer.dimy + j;
				break;
			default:
				idn += j * layer.dimz + k;
			}
			//int idn = i * layer.dimy * layer.dimz + j * layer.dimz + k;
			if (nodes[idn] == NODE_IN)
			switch (var)
			{
			case type_U: temp.elem(temp.u, i, j, k) = (temp.elem(temp.u, i, j, k) +  get(x,t) ) / 2; break;
			case type_V: temp.elem(temp.v, i, j, k) = (temp.elem(temp.v, i, j, k) +  get(x,t) ) / 2; break;
			case type_W: temp.elem(temp.w, i, j, k) = (temp.elem(temp.w, i, j, k) +  get(x,t) ) / 2; break;
			case type_T: temp.elem(temp.T, i, j, k) = (temp.elem(temp.T, i, j, k) +  get(x,t) ) / 2; break;
			}
#endif

			switch (dir)
			{
			case X: i++; break;
			case Y: j++; break;
			case Z: k++; break;
			case Z_as_Y: j++; break;
			}
		}
	}

	template<int dir, int var, int swipe>
	__global__ void solve_segments( FluidParamsGPU p, int num_seg, Segment3D *segs, NodesBoundary3D *nodesBounds, NodeType *nodeTypes, TimeLayer3D_GPU cur, TimeLayer3D_GPU temp, TimeLayer3D_GPU next,
									FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int  max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		NodesBoundary3D &nodes = nodesBounds[id];

		int n = seg.size;

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}

		switch (swipe)
		{
		case ALL:
			apply_bc0<dir, var>(seg.posx, seg.posy, seg.posz, cur.dimy, cur.dimz, get(b,0), get(c,0), get(d,0), nodes.first, seg.type);
			apply_bc1<dir, var>(seg.endx, seg.endy, seg.endz, cur.dimy, cur.dimz, get(a,n-1), get(b,n-1), get(d,n-1), nodes.last, seg.type);
		case FORWARD:
			build_matrix<dir, var>(p, seg.posx, seg.posy, seg.posz, a, b, c, d, n, cur, temp, id, num_seg, max_n_max_n, seg.type);
		case BACK:			
			solve_tridiagonal<dir, swipe>(a, b, c, d, x, n, id, num_seg, max_n_max_n, dimX, seg.type);
			break;
		}
			
		switch (swipe)
		{
		case ALL:
		case BACK:
			update_segment<dir, var>(x, seg, nodeTypes, temp, next, id, num_seg, max_n_max_n);
			break;
		}
	}

	template<int dir, int var>
	__global__ void update_segments(int num_seg, Segment3D *segs, NodeType *nodeTypes, TimeLayer3D_GPU temp, TimeLayer3D_GPU next,  FTYPE *x, int  max_n_max_n, int id_shift = 0)
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		//NodeType &nodes = nodesBounds[id];

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}
		update_segment<dir, var>(x, seg, nodeTypes, temp, next, id, num_seg, max_n_max_n);
	}

	template<int dir, int var>
	__global__ void build_matrix( FluidParamsGPU p, int num_seg, Segment3D *segs, NodesBoundary3D *nodesBounds, TimeLayer3D_GPU cur, TimeLayer3D_GPU temp, 
								  FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, int max_n_max_n, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		NodesBoundary3D &nodes = nodesBounds[id];

		int n = seg.size;

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}
		
		apply_bc0<dir, var>(seg.posx, seg.posy, seg.posz, cur.dimy, cur.dimz, get(b,0), get(c,0), get(d,0), nodes.first, seg.type);
		apply_bc1<dir, var>(seg.endx, seg.endy, seg.endz, cur.dimy, cur.dimz, get(a,n-1), get(b,n-1), get(d,n-1), nodes.last, seg.type);
		build_matrix<dir, var>(p, seg.posx, seg.posy, seg.posz, a, b, c, d, n, cur, temp, id, num_seg, max_n_max_n, seg.type);		
	}

	template<int dir, int var, int swipe>
	__global__ void solve_matrix( int num_seg, Segment3D *segs, NodeType *nodeTypes, TimeLayer3D_GPU temp, TimeLayer3D_GPU next, 
								  FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		//NodesBoundary3D &nodes = nodesBounds[id];

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}

		int n = seg.size;

		solve_tridiagonal<dir, swipe>(a, b, c, d, x, n, id, num_seg, max_n_max_n, dimX, seg.type);
		
		switch (swipe)
		{
		case ALL:
			update_segment<dir, var>(x, seg, nodeTypes, temp, next, id, num_seg, max_n_max_n);
			break;
		}
	}

	template<DirType dir, VarType var>
	void LaunchSolveSegments_dir_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	/*
		Y and Z direction only (and X if nGPUs = 1)
	*/
	{
		GPUplan *pGPUplan = GPUplan::Instance();

		int max_n_max_n;
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );
		dim3 block(SOLVER_BLOCK_DIM);

		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			int dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = pGPUplan->node(i)->getLength1D() * max_n;  // valid for Y and Z direction only if nGPUs > 1

			switch( decomposeOpt )
			{
			case true:
				build_matrix<dir, var><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], cur, temp, d_a[i], d_b[i], d_c[i], d_d[i], max_n_max_n );
				break;

			case false:
				//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);
				solve_segments<dir, var, ALL><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], nodeTypes[i], cur, temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
				break;
			}
		}

		if ( decomposeOpt )
		{
			for (int i = 0; i < pGPUplan->size(); i++)
			{
				pGPUplan->setDevice(i);
				cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

				int dimX = pGPUplan->node(i)->getLength1D();
				dim3 grid((num_seg[i] + block.x - 1)/block.x);

				max_n_max_n = pGPUplan->node(i)->getLength1D() * max_n;

				solve_matrix<dir, var, ALL><<<grid, block>>>( num_seg[i], segs[i], nodeTypes[i], temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], dimX, max_n_max_n );
			}
		}
		pGPUplan->deviceSynchronize();
	}

	template<VarType var>
	void LaunchSolveSegments_X_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt, int numSegs, FTYPE *mpi_buf = NULL )
	/*
		X direction only
	*/
	{
		GPUplan *pGPUplan = GPUplan::Instance();
		PARAplan* pplan = PARAplan::Instance();
		int irank =  pplan->rank();
		int size = pplan->size();
		
		if (pGPUplan->size() == 1 && pplan->size() == 1)
		{
			LaunchSolveSegments_dir_var<X, var>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt );
			return;
		}
		
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );
		int max_n_max_n = max_n * max_n;

		int haloSize = max_n * max_n * MAX_SEGS_PER_ROW;
		int comSize = numSegs;
		int dimX;

		dim3 block(SOLVER_BLOCK_DIM);

		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);
			dim3 grid((num_seg[i] + block.x - 1)/block.x);
			build_matrix<X, var><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], cur, temp, d_a[i], d_b[i], d_c[i], d_d[i], max_n_max_n );
		}
#ifdef __PARA
		if (pplan->size() > 1)
		{			
			pGPUplan->setDevice(0);
			paraDevRecv<FTYPE, FORWARD>(d_c[0], mpi_buf, comSize, 666);
			paraDevRecv<FTYPE, FORWARD>(d_d[0], mpi_buf + comSize, comSize, 667);
		}
#endif
		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);
			solve_matrix<X, var, FORWARD><<<grid, block>>>( num_seg[i], segs[i], nodeTypes[i], temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i < pGPUplan->size() - 1) //  send to node n+1
			{
				haloMemcpyPeer<FTYPE, FORWARD>( d_c, i, haloSize, dimX * haloSize, 0, comSize);
				haloMemcpyPeer<FTYPE, FORWARD>( d_d, i, haloSize, dimX * haloSize, 0, comSize);
			}
		}
#ifdef __PARA
		if (pplan->size() > 0)
		{
			pGPUplan->setDevice(pGPUplan->size()-1);
			paraDevSend<FTYPE, FORWARD>(d_c[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf, comSize, 666);
			paraDevSend<FTYPE, FORWARD>(d_d[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf + comSize, comSize, 667);
			paraDevRecv<FTYPE, BACK>(d_x[pGPUplan->size()-1] + haloSize +  dimX * haloSize, mpi_buf, comSize, 668);
		}
#endif
		for (int i = pGPUplan->size() - 1; i >= 0; i--)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = max_n * max_n;
			solve_matrix<X, var, BACK><<<grid, block>>>( num_seg[i], segs[i], nodeTypes[i], temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i > 0)
				haloMemcpyPeer<FTYPE, BACK>(d_x, i, haloSize, pGPUplan->node(i-1)->getLength1D()*haloSize, 0, comSize);
		}
#ifdef __PARA		
		if (pplan->size() > 0)
		{			
			pGPUplan->setDevice(0);
			paraDevSend<FTYPE, BACK>(d_x[0] + haloSize, mpi_buf, comSize, 668);
		}
#endif
		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);
			dim3 grid((num_seg[i] + block.x - 1)/block.x);
			update_segments<X, var><<<grid, block>>>( num_seg[i], segs[i], nodeTypes[i], temp, next, d_x[i],  max_n_max_n);
		}
		pGPUplan->deviceSynchronize();
	}

	template<DirType dir>
	void LaunchSolveSegments_dir( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_dir_var<dir, type_U>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_V: LaunchSolveSegments_dir_var<dir, type_V>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_W: LaunchSolveSegments_dir_var<dir, type_W>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_T: LaunchSolveSegments_dir_var<dir, type_T>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		}
	}

	void LaunchSolveSegments_X( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt, int numSegs, FTYPE *mpi_buf = NULL )
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_X_var<type_U>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt, numSegs, mpi_buf ); break;
		case type_V: LaunchSolveSegments_X_var<type_V>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt, numSegs, mpi_buf ); break;
		case type_W: LaunchSolveSegments_X_var<type_W>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt, numSegs, mpi_buf ); break;
		case type_T: LaunchSolveSegments_X_var<type_T>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt, numSegs, mpi_buf ); break;
		}
	}

	void SolveSegments_GPU( FTYPE dt, FluidParams params, int *num_seg, Segment3D **segs, VarType var, DirType dir, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D *cur, TimeLayer3D *temp, TimeLayer3D *next,
							FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt,  int numSegs, FTYPE *mpi_buf )
	{
		TimeLayer3D_GPU d_cur( cur );
		TimeLayer3D_GPU d_temp( temp );
		TimeLayer3D_GPU d_next( next );

		FluidParamsGPU p( var, dir, dt, cur->dx, cur->dy, cur->dz, params );

		switch( dir )
		{
		case X: LaunchSolveSegments_X( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt, numSegs, mpi_buf ); break;
		case Y: LaunchSolveSegments_dir<Y>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case Z: LaunchSolveSegments_dir<Z>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case Z_as_Y: LaunchSolveSegments_dir<Z_as_Y>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		}
	}
}
