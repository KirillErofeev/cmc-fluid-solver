#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2011 Nikolai Sakharnykh
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "AdiSolver3D.h"

#define SEG_BLOCK_DIM_X		32
#define SEG_BLOCK_DIM_Y		8

#if( __CUDA_ARCH__ < 120 )
#define SOLVER_BLOCK_DIM	128
#else
#define SOLVER_BLOCK_DIM	256
#endif

namespace FluidSolver3D
{
	struct FluidParamsGPU
	{
		FTYPE vis_dx2;
		FTYPE dt, dx, dy, dz;
		FTYPE v_T, t_phi;

		FluidParamsGPU( VarType var, DirType dir, FTYPE _dt, FTYPE _dx, FTYPE _dy, FTYPE _dz, FluidParams _params ) : 
			dt(_dt), dx(_dx), dy(_dy), dz(_dz)
		{
			switch (var)
			{
			case type_U:
			case type_V:
			case type_W:	
				switch (dir)
				{
				case X:	vis_dx2 = _params.v_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.v_vis / (dy * dy); break;
				case Z: case Z_as_Y: vis_dx2 = _params.v_vis / (dz * dz); break;
				}
				break;
			case type_T:
				switch (dir)
				{
				case X:	vis_dx2 = _params.t_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.t_vis / (dy * dy); break;
				case Z: case Z_as_Y:  vis_dx2 = _params.t_vis / (dz * dz); break;
				}
				break;
			}
						
			v_T = _params.v_T;
			t_phi = _params.t_phi;
		}
	};

#if 1
	// interleave matrix arrays for better memory access
	#define get(a, elem_id)			a[id + (elem_id + 1) * max_n_max_n * MAX_SEGS_PER_ROW]
#else
	// sequential layout - bad access pattern,  currently not implemented for MGPU
//	#define get(a, elem_id)			a[elem_id + id * max_n]
#endif

template<int dir, int swipe, int var>
	__device__ void solve_tridiagonal(FluidParamsGPU params, FTYPE *c, FTYPE *d, FTYPE *x, Segment3D &seg, int num, NodesBoundary3D &nodesBound, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, int id, int num_seg, int max_n_max_n, int dimX, SegmentType segType = BOUND)
	{
		FTYPE a_val, b_val, c_val, d_val;
		int i = seg.posx;
		int j = seg.posy;
		int k = seg.posz;

		switch (swipe)
		{
			case ALL:
			case FORWARD:
			{
				// apply boundary 0 conditions:
				switch (dir)
				{
				case X:
					if (segType == UNBOUND || segType == BOUND_END)
						break;
				default:
					if ((var == type_T && nodesBound.first.bc_temp == BC_FREE) ||
						(var != type_T && nodesBound.first.bc_vel == BC_FREE))
					{
						// free: f(0) = 2 * f(1) - f(2)
						b_val = 2.0; 
						c_val = -1.0; 
						d_val = 0.0;
					}
					else
					{
						// no-slip: f(0) = f(1)
						b_val = 1.0; 
						c_val = 0.0; 

						switch (var)
						{
							case type_U: d_val = nodesBound.first.v.x; break;  
							case type_V: d_val = nodesBound.first.v.y; break;
							case type_W: d_val = nodesBound.first.v.z; break;
							case type_T: d_val = nodesBound.first.T; break;
						}
					}
				}

				int start = 1;
				int end = num - 1;

				switch (dir)
				{
				case X:
						switch (segType)
						{
						case UNBOUND:
							start = 0;
							end = num;
							break;
						case BOUND_END:
							start = 0;
							get(c, num-1) = 0.0;
							break;
						case BOUND:
							get(c, num-1) = 0.0;
						case BOUND_START:
							get(c,0) = c_val / b_val;
							get(d,0) = d_val / b_val;
							end = num;
							break;
						}
						break;
				default:
					get(c,0) = c_val / b_val;
					get(d,0) = d_val / b_val;
					get(c,num-1) = 0.0;
					break;
				}

				for (int p = start; p < end; p++)
				{
					// Build matrix:
					switch (dir)
					{
					case X:		
						a_val = - temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
						b_val = 3 / params.dt  +  2 * params.vis_dx2; 
						c_val = temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
				
						switch (var)	
						{
						case type_U: d_val = cur.elem(cur.u, i+p, j, k) * 3 / params.dt - params.v_T * temp.d_x(temp.T, params.dx, i+p, j, k); break;
						case type_V: d_val = cur.elem(cur.v, i+p, j, k) * 3 / params.dt; break;
						case type_W: d_val = cur.elem(cur.w, i+p, j, k) * 3 / params.dt; break;
						case type_T: d_val = cur.elem(cur.T, i+p, j, k) * 3 / params.dt + params.t_phi * temp.DissFuncX(params.dx, params.dy, params.dz, i+p, j, k); break;
						}	
						break;

					case Y:
						a_val = - temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
						b_val = 3 / params.dt  +  2 * params.vis_dx2; 
						c_val = temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
				
						switch (var)	
						{
						case type_U: d_val = cur.elem(cur.u, i, j+p, k) * 3 / params.dt; break;
						case type_V: d_val = cur.elem(cur.v, i, j+p, k) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dy, i, j+p, k); break;
						case type_W: d_val = cur.elem(cur.w, i, j+p, k) * 3 / params.dt; break;
						case type_T: d_val = cur.elem(cur.T, i, j+p, k) * 3 / params.dt + params.t_phi * temp.DissFuncY(params.dx, params.dy, params.dz, i, j+p, k); break;
						}
						break;

					case Z:
						a_val = - temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
						b_val = 3 / params.dt  +  2 * params.vis_dx2; 
						c_val = temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
				
						switch (var)	
						{
						case type_U: d_val = cur.elem(cur.u, i, j, k+p) * 3 / params.dt; break;
						case type_V: d_val = cur.elem(cur.v, i, j, k+p) * 3 / params.dt; break;
						case type_W: d_val = cur.elem(cur.w, i, j, k+p) * 3 / params.dt - params.v_T * temp.d_z(temp.T, params.dz, i, j, k+p); break;
						case type_T: d_val = cur.elem(cur.T, i, j, k+p) * 3 / params.dt + params.t_phi * temp.DissFuncZ(params.dx, params.dy, params.dz, i, j, k+p); break;
						}
						break;

					case Z_as_Y:
						a_val = - temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
						b_val = 3 / params.dt  +  2 * params.vis_dx2; 
						c_val = temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
				
						switch (var)	
						{
						case type_U: d_val = cur.elem(cur.u, i, k+p, j) * 3 / params.dt; break;
						case type_V: d_val = cur.elem(cur.v, i, k+p, j) * 3 / params.dt; break;
						case type_W: d_val = cur.elem(cur.w, i, k+p, j) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dz, i, k+p, j); break;
						case type_T: d_val = cur.elem(cur.T, i, k+p, j) * 3 / params.dt + params.t_phi * temp.DissFuncZ_as_Y(params.dx, params.dz, params.dy, i, k+p, j); break;
						}
						break;
					} // end of build matrix

					// forward solver step:
					get(c,p) = c_val / (b_val - a_val * get(c,p-1));
					get(d,p) = (d_val - get(d,p-1) * a_val) / (b_val - a_val * get(c,p-1));
				}

				// apply boundary 1 conditions:
				switch (dir)
				{
				case X:
					if (segType == UNBOUND || segType == BOUND_START)
						break;
				default:
					if ((var == type_T && nodesBound.last.bc_temp == BC_FREE) ||
						(var != type_T && nodesBound.last.bc_vel == BC_FREE))
					{
						// free: f(N) = 2 * f(N-1) - f(N-2)
						a_val = -1.0;
						b_val = 2.0;
						d_val = 0.0;
					}
					else
					{
						// no-slip: f(N) = f(N-1)
						a_val = 0.0; 
						b_val = 1.0; 

						switch (var)
						{
							case type_U: d_val = nodesBound.last.v.x; break;
							case type_V: d_val = nodesBound.last.v.y; break;
							case type_W: d_val = nodesBound.last.v.z; break;
							case type_T: d_val = nodesBound.last.T; break;
						}
					}
					// move boundary setting in here
					get(c, num-1) = 0.0;
					get(d,num-1) = (d_val - get(d,num-2) * a_val) / (b_val - a_val * get(c,num-2));
				}

				switch (dir)
				{
				case X:
					// copy segments' ends to halo area
					get(c, dimX-1) = get(c, num-1);
					get(d, dimX-1) = get(d, num-1);
					break;
				}

				break; // SWIPE
			}			
		}

		switch (swipe)
		{
			case ALL:
			case BACK:
			{
				int end = num - 1;

				switch (dir)
				{
				case X:
					switch (segType)
					{
					case UNBOUND:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_START:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_END:
						get(x, num-1) = get(d,num-1);
						break;
					case BOUND:
						get(x, num-1) = get(d,num-1);
						break;
					}
					break;
				default:
					get(x, num-1) = get(d,num-1);
					break;
				}

				for (int i = end-1; i >= 0; i--) 
					get(x,i) = get(d,i) -  get(c,i) * get(x, i+1);
				break;
			}
		}
	}

	template<int dir, int var>
	__device__ void update_segment( FTYPE *x, Segment3D &seg, NodeType *nodes, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &layer, int id, int num_seg, int max_n_max_n )
	{
		int i = seg.posx;
		int j = seg.posy;
		int k = seg.posz;

		if( dir == Z_as_Y ) 
		{
			k = seg.posy;
			j = seg.posz;
		}

		for (int t = 0; t < seg.size; t++)
		{
			switch (var)
			{
			case type_U: layer.elem(layer.u, i, j, k) = get(x,t); break;
			case type_V: layer.elem(layer.v, i, j, k) = get(x,t); break;
			case type_W: layer.elem(layer.w, i, j, k) = get(x,t); break;
			case type_T: layer.elem(layer.T, i, j, k) = get(x,t); break;
			}
#if INTERNAL_MERGE_ENABLE == 1
			//if (t == 0)
			//	if (nodes.first.type != NODE_IN)
			//		continue;
			//if (t == seg.size-1)
			//	if (nodes.last.type != NODE_IN)
			//		continue;

			int idn;
			idn = i * layer.dimy * layer.dimz;
			switch(dir)
			{
			case Z_as_Y:
				idn += k * layer.dimy + j;
				break;
			default:
				idn += j * layer.dimz + k;
			}
			//int idn = i * layer.dimy * layer.dimz + j * layer.dimz + k;
			if (nodes[idn] == NODE_IN)
			switch (var)
			{
			case type_U: temp.elem(temp.u, i, j, k) = (temp.elem(temp.u, i, j, k) +  get(x,t) ) / 2; break;
			case type_V: temp.elem(temp.v, i, j, k) = (temp.elem(temp.v, i, j, k) +  get(x,t) ) / 2; break;
			case type_W: temp.elem(temp.w, i, j, k) = (temp.elem(temp.w, i, j, k) +  get(x,t) ) / 2; break;
			case type_T: temp.elem(temp.T, i, j, k) = (temp.elem(temp.T, i, j, k) +  get(x,t) ) / 2; break;
			}
#endif

			switch (dir)
			{
			case X: i++; break;
			case Y: j++; break;
			case Z: k++; break;
			case Z_as_Y: j++; break;
			}
		}
	}

	template<int dir, int var, int swipe>
	__global__ void solve_segments( FluidParamsGPU p, int num_seg, Segment3D *segs, NodesBoundary3D *nodesBounds, NodeType *nodeTypes, TimeLayer3D_GPU cur, TimeLayer3D_GPU temp, TimeLayer3D_GPU next,
									FTYPE *c, FTYPE *d, FTYPE *x, int  max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		NodesBoundary3D &nodes = nodesBounds[id];

		int n = seg.size;

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}

		solve_tridiagonal<dir, swipe, var>(p, c, d, x, seg, n, nodes, cur, temp, id, num_seg, max_n_max_n, dimX, seg.type);
			
		switch (swipe)
		{
		case ALL:
		case BACK:
			update_segment<dir, var>(x, seg, nodeTypes, temp, next, id, num_seg, max_n_max_n);
			break;
		}
	}

	template<int dir, int var>
	__global__ void update_segments(int num_seg, Segment3D *segs, NodeType *nodeTypes, TimeLayer3D_GPU temp, TimeLayer3D_GPU next,  FTYPE *x, int  max_n_max_n, int id_shift = 0)
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];
		//NodeType &nodes = nodesBounds[id];

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}
		update_segment<dir, var>(x, seg, nodeTypes, temp, next, id, num_seg, max_n_max_n);
	}

	template<DirType dir, VarType var>
	void LaunchSolveSegments_dir_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_c, FTYPE **d_d, FTYPE **d_x )
// Y or Z direction only (and X if nGPUs = 1)
	{
		GPUplan *pGPUplan = GPUplan::Instance();

		int max_n_max_n;
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );
		dim3 block(SOLVER_BLOCK_DIM);

		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			int dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = pGPUplan->node(i)->getLength1D() * max_n;  // valid for Y and Z direction only if nGPUs > 1

			//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);			
			solve_segments<dir, var, ALL><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], nodeTypes[i], cur, temp, next, d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
		}
		pGPUplan->deviceSynchronize();
	}

	template<VarType var>
	void LaunchSolveSegments_X_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, int numSegs, FTYPE *mpi_buf = NULL )
	/*
		X direction only
	*/
	{
		GPUplan *pGPUplan = GPUplan::Instance();
		PARAplan* pplan = PARAplan::Instance();
		int irank =  pplan->rank();
		int size = pplan->size();
		
		if (pGPUplan->size() == 1 && pplan->size() == 1)
		{
			LaunchSolveSegments_dir_var<X, var>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x );
			return;
		}
		
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );
		int max_n_max_n = max_n * max_n;

		int haloSize = max_n * max_n * MAX_SEGS_PER_ROW;
		int comSize = numSegs;
		int dimX;

		dim3 block(SOLVER_BLOCK_DIM);

#ifdef __PARA
		if (pplan->size() > 1)
		{			
			pGPUplan->setDevice(0);
			paraDevRecv<FTYPE, FORWARD>(d_c[0], mpi_buf, comSize, 666);
			paraDevRecv<FTYPE, FORWARD>(d_d[0], mpi_buf + comSize, comSize, 667);
		}
#endif
		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);
			solve_segments<X, var, FORWARD><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], nodeTypes[i], cur, temp, next, d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i < pGPUplan->size() - 1) //  send to node n+1
			{
				haloMemcpyPeer<FTYPE, FORWARD>( d_c, i, haloSize, dimX * haloSize, 0, comSize);
				haloMemcpyPeer<FTYPE, FORWARD>( d_d, i, haloSize, dimX * haloSize, 0, comSize);
			}
		}
#ifdef __PARA
		if (pplan->size() > 0)
		{
			pGPUplan->setDevice(pGPUplan->size()-1);
			paraDevSend<FTYPE, FORWARD>(d_c[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf, comSize, 666);
			paraDevSend<FTYPE, FORWARD>(d_d[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf + comSize, comSize, 667);
			paraDevRecv<FTYPE, BACK>(d_x[pGPUplan->size()-1] + haloSize +  dimX * haloSize, mpi_buf, comSize, 668);
		}
#endif
		for (int i = pGPUplan->size() - 1; i >= 0; i--)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = max_n * max_n;
			solve_segments<X, var, BACK><<<grid, block>>>( p, num_seg[i], segs[i], nodesBounds[i], nodeTypes[i], cur, temp, next, d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i > 0)
				haloMemcpyPeer<FTYPE, BACK>(d_x, i, haloSize, pGPUplan->node(i-1)->getLength1D()*haloSize, 0, comSize);
		}
#ifdef __PARA		
		if (pplan->size() > 0)
		{			
			pGPUplan->setDevice(0);
			paraDevSend<FTYPE, BACK>(d_x[0] + haloSize, mpi_buf, comSize, 668);
		}
#endif
		for (int i = 0; i < pGPUplan->size(); i++)
		{
			pGPUplan->setDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);
			dim3 grid((num_seg[i] + block.x - 1)/block.x);
			update_segments<X, var><<<grid, block>>>( num_seg[i], segs[i], nodeTypes[i], temp, next, d_x[i],  max_n_max_n);
		}
		pGPUplan->deviceSynchronize();
	}

	template<DirType dir>
	void LaunchSolveSegments_dir( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_c, FTYPE **d_d, FTYPE **d_x )
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_dir_var<dir, type_U>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x ); break;
		case type_V: LaunchSolveSegments_dir_var<dir, type_V>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x ); break;
		case type_W: LaunchSolveSegments_dir_var<dir, type_W>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x ); break;
		case type_T: LaunchSolveSegments_dir_var<dir, type_T>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x ); break;
		}
	}

	void LaunchSolveSegments_X( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, int numSegs, FTYPE *mpi_buf = NULL )
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_X_var<type_U>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x, numSegs, mpi_buf ); break;
		case type_V: LaunchSolveSegments_X_var<type_V>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x, numSegs, mpi_buf ); break;
		case type_W: LaunchSolveSegments_X_var<type_W>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x, numSegs, mpi_buf ); break;
		case type_T: LaunchSolveSegments_X_var<type_T>( p, num_seg, segs, nodesBounds, nodeTypes, cur, temp, next, d_c, d_d, d_x, numSegs, mpi_buf ); break;
		}
	}

	void SolveSegments_GPU( FTYPE dt, FluidParams params, int *num_seg, Segment3D **segs, VarType var, DirType dir, NodesBoundary3D **nodesBounds, NodeType **nodeTypes, TimeLayer3D *cur, TimeLayer3D *temp, TimeLayer3D *next,
							FTYPE **d_c, FTYPE **d_d, FTYPE **d_x,  int numSegs, FTYPE *mpi_buf )
	{
		TimeLayer3D_GPU d_cur( cur );
		TimeLayer3D_GPU d_temp( temp );
		TimeLayer3D_GPU d_next( next );

		FluidParamsGPU p( var, dir, dt, cur->dx, cur->dy, cur->dz, params );

		switch( dir )
		{
		case X: LaunchSolveSegments_X( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_c, d_d, d_x, numSegs, mpi_buf ); break;
		case Y: LaunchSolveSegments_dir<Y>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_c, d_d, d_x ); break;
		case Z: LaunchSolveSegments_dir<Z>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_c, d_d, d_x ); break;
		case Z_as_Y: LaunchSolveSegments_dir<Z_as_Y>( p, num_seg, segs, var, nodesBounds, nodeTypes, d_cur, d_temp, d_next, d_c, d_d, d_x ); break;
		}
	}
}
