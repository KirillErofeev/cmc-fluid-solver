#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2011 Nikolai Sakharnykh
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "AdiSolver3D.h"

#if( __CUDA_ARCH__ < 120 )
#define SOLVER_BLOCK_DIM	128
#else
#define SOLVER_BLOCK_DIM	256
#endif

#define SEG_BLOCK_DIM_X		32
#define SEG_BLOCK_DIM_Y		8

#include "../Common/test_util.h"

namespace FluidSolver3D
{
	struct FluidParamsGPU
	{
		FTYPE vis_dx2;
		FTYPE dt, dx, dy, dz;
		FTYPE v_T, t_phi;

		FluidParamsGPU( VarType var, DirType dir, FTYPE _dt, FTYPE _dx, FTYPE _dy, FTYPE _dz, FluidParams _params ) : 
			dt(_dt), dx(_dx), dy(_dy), dz(_dz)
		{
			switch (var)
			{
			case type_U:
			case type_V:
			case type_W:	
				switch (dir)
				{
				case X:	vis_dx2 = _params.v_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.v_vis / (dy * dy); break;
				case Z: case Z_as_Y: vis_dx2 = _params.v_vis / (dz * dz); break;
				}
				break;
			case type_T:
				switch (dir)
				{
				case X:	vis_dx2 = _params.t_vis / (dx * dx); break;
				case Y: vis_dx2 = _params.t_vis / (dy * dy); break;
				case Z: case Z_as_Y:  vis_dx2 = _params.t_vis / (dz * dz); break;
				}
				break;
			}
						
			v_T = _params.v_T;
			t_phi = _params.t_phi;
		}
	};

#if 1
	// interleave matrix arrays for better memory access
	#define get(a, elem_id)			a[id + (elem_id + 1) * max_n_max_n * MAX_SEGS_PER_ROW]
#else
	// sequential layout - bad access pattern,  currently not implemented for MGPU
//	#define get(a, elem_id)			a[elem_id + id * max_n]
#endif

	template<int dir, int var>
	__device__ void apply_bc0(int i, int j, int k, int dimy, int dimz, FTYPE &b0, FTYPE &c0, FTYPE &d0, Node *nodes, SegmentType segType = BOUND)
	{
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				return;
			case BOUND_END:
				return;
			};
		}

		//b0 = c0 = d0 = 1.;
	//	return;

		int id = i * dimy * dimz;
		if (dir != Z_as_Y) id += j * dimz + k;
			else id += j * dimy + k;

		if ((var == type_T && nodes[id].bc_temp == BC_FREE) ||
			(var != type_T && nodes[id].bc_vel == BC_FREE))
		{
			// free: f(0) = 2 * f(1) - f(2)
			b0 = 2.0; 
			c0 = -1.0; 
			d0 = 0.0; 
		}
		else
		{
			// no-slip: f(0) = f(1)
			b0 = 1.0; 
			c0 = 0.0; 
			switch (var)
			{
			case type_U: d0 = nodes[id].v.x; break;
			case type_V: d0 = nodes[id].v.y; break;
			case type_W: d0 = nodes[id].v.z; break;
			case type_T: d0 = nodes[id].T; break;
			}
		}
	}

	template<int dir, int var>
	__device__ void apply_bc1(int i, int j, int k, int dimy, int dimz, FTYPE &a1, FTYPE &b1, FTYPE &d1, Node *nodes, SegmentType segType = BOUND)
	{
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				return;
			case BOUND_START:
				return;
			}
		}

		//a1 = b1 = d1 = 2.;
		//return;

		int id = i * dimy * dimz;
		if (dir != Z_as_Y) id += j * dimz + k;
			else id += j * dimy + k;

		if ((var == type_T && nodes[id].bc_temp == BC_FREE) ||
			(var != type_T && nodes[id].bc_vel == BC_FREE))
		{
			// free: f(N) = 2 * f(N-1) - f(N-2)
			a1 = -1.0; 
			b1 = 2.0; 
			d1 = 0.0;
		}
		else
		{
			// no-slip: f(N) = f(N-1)
			a1 = 0.0; 
			b1 = 1.0; 
			switch (var)
			{
			case type_U: d1 = nodes[id].v.x; break;
			case type_V: d1 = nodes[id].v.y; break;
			case type_W: d1 = nodes[id].v.z; break;
			case type_T: d1 = nodes[id].T; break;
			}
		}
	}	

	template<int dir, int var>
	__device__ void build_matrix(FluidParamsGPU params, int i, int j, int k, FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, int n, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, int id, int num_seg, int max_n_max_n, int dimX, SegmentType segType = BOUND)
	{	
		int ps;
		int start = 1;
		int end = n-1;
		switch (dir)
		{
		case X:
			switch (segType)
			{
			case UNBOUND:
				start = 0; end = n;				
				break;
			case BOUND_START:
				end = n;
				break;
			case BOUND_END:
				start = 0;
				break;
			}
		}

		for (int p = start; p < end; p++)
		{
			switch (dir)
			{
			case X:		
				ps = dimX - n + p; // allign segments to the end
				get(a,ps) = - temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
				get(b,ps) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,ps) = temp.elem(temp.u, i+p, j, k) / (2 * params.dx) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,ps) = cur.elem(cur.u, i+p, j, k) * 3 / params.dt - params.v_T * temp.d_x(temp.T, params.dx, i+p, j, k); break;
				case type_V: get(d,ps) = cur.elem(cur.v, i+p, j, k) * 3 / params.dt; break;
				case type_W: get(d,ps) = cur.elem(cur.w, i+p, j, k) * 3 / params.dt; break;
				case type_T: get(d,ps) = cur.elem(cur.T, i+p, j, k) * 3 / params.dt + params.t_phi * temp.DissFuncX(params.dx, params.dy, params.dz, i+p, j, k); break;
				}	
				break;

			case Y:
				get(a,p) = - temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.v, i, j+p, k) / (2 * params.dy) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, j+p, k) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, j+p, k) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dy, i, j+p, k); break;
				case type_W: get(d,p) = cur.elem(cur.w, i, j+p, k) * 3 / params.dt; break;
				case type_T: get(d,p) = cur.elem(cur.T, i, j+p, k) * 3 / params.dt + params.t_phi * temp.DissFuncY(params.dx, params.dy, params.dz, i, j+p, k); break;
				}
				break;

			case Z:
				get(a,p) = - temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.w, i, j, k+p) / (2 * params.dz) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, j, k+p) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, j, k+p) * 3 / params.dt; break;
				case type_W: get(d,p) = cur.elem(cur.w, i, j, k+p) * 3 / params.dt - params.v_T * temp.d_z(temp.T, params.dz, i, j, k+p); break;
				case type_T: get(d,p) = cur.elem(cur.T, i, j, k+p) * 3 / params.dt + params.t_phi * temp.DissFuncZ(params.dx, params.dy, params.dz, i, j, k+p); break;
				}
				break;

			case Z_as_Y:
				get(a,p) = - temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
				get(b,p) = 3 / params.dt  +  2 * params.vis_dx2; 
				get(c,p) = temp.elem(temp.w, i, k+p, j) / (2 * params.dz) - params.vis_dx2; 
				
				switch (var)	
				{
				case type_U: get(d,p) = cur.elem(cur.u, i, k+p, j) * 3 / params.dt; break;
				case type_V: get(d,p) = cur.elem(cur.v, i, k+p, j) * 3 / params.dt; break;
				case type_W: get(d,p) = cur.elem(cur.w, i, k+p, j) * 3 / params.dt - params.v_T * temp.d_y(temp.T, params.dz, i, k+p, j); break;
				case type_T: get(d,p) = cur.elem(cur.T, i, k+p, j) * 3 / params.dt + params.t_phi * temp.DissFuncZ_as_Y(params.dx, params.dz, params.dy, i, k+p, j); break;
				}
				break;
			}
		}
	}

template<int dir, int swipe>
	__device__ void solve_tridiagonal(FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int num, int id, int num_seg, int max_n_max_n, int dimX, SegmentType segType = BOUND)
	{
		switch (swipe)
		{
			case ALL:
			case FORWARD:
			{
				int start = 1;

				switch (dir)
				{
				case X:
						switch (segType)
						{
						case UNBOUND:
							start = 0;
							get(c,dimX-num-1) = get(c,-1);
							get(d,dimX-num-1) = get(d,-1);
							break;
						case BOUND_END:
							start = 0;
							get(c,dimX-num-1) = get(c,-1);
							get(d,dimX-num-1) = get(d,-1);
							get(c, dimX-1) = 0.0;
							break;
						case BOUND:
							get(c,dimX-1) = 0.0;
							get(c,dimX-num) = get(c,dimX-num) / get(b,dimX-num);
							get(d,dimX-num) = get(d,dimX-num) / get(b,dimX-num);
							break;
						case BOUND_START:
							get(c,dimX-num) = get(c,dimX-num) / get(b,dimX-num);
							get(d,dimX-num) = get(d,dimX-num) / get(b,dimX-num);
							break;
						}
						break;
				default:
					get(c,0) = get(c,0) / get(b,0);
					get(d,0) = get(d,0) / get(b,0);
					get(c,num-1) = 0.0;
					break;
				}

				int is;
				for (int i = start; i < num; i++)
				{
					switch (dir)
					{
					case X:
						is = dimX - num + i;
						break;
					default:
						is = i;
						break;
					}
					get(c,is) = get(c,is) / (get(b,is) - get(a,is) * get(c,is-1));
					get(d,is) = (get(d,is) - get(d,is-1) * get(a,is)) / (get(b,is) - get(a,is) * get(c,is-1));
					/*
					switch (segType)
					{
						case BOUND_START:
							if ( i ==  num - 1)
							{
								printf("BOUND_START: id = %d    get(d, %d) = %f\n",id ,is, get(d, is));
							}
							break;
						case BOUND_END:
							if ( i == start)
								printf("BOUND_END: id = %d    get(d, %d) = %f\n",id ,is, get(d, is));
							break;
					}
					/**/
				}
				break;
			}			
		}

		switch (swipe)
		{
			case ALL:
			case BACK:
			{
				int end = num - 1;

				switch (dir)
				{
				case X:
					switch (segType)
					{
					case UNBOUND:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_START:
						end = num;
						get(x,num) = get(x,dimX);
						break;
					case BOUND_END:
						get(x, num-1) = get(d,dimX-1);
						break;
					case BOUND:
						get(x, num-1) = get(d,dimX-1);
						break;
					}
					break;
				default:
					get(x, num-1) = get(d,num-1);
					break;
				}

				int is;
				for (int i = end-1; i >= 0; i--) 
				{
					switch (dir)
					{
					case X:
						is = dimX - num + i;
						break;
					default:
						is = i;
					}
					get(x,i) = get(d,is) -  get(c,is) * get(x, i+1);
					/*
					switch (segType)
					{
						case BOUND_START:
							if ( i ==  end - 1)
							{
								printf("BOUND_START: id = %d    get(x, %d+1) = %f\n",id ,i, get(x, i+1));
							}
							break;
						case BOUND_END:
							if ( i == 0)
								printf("BOUND_END: id = %d    get(x, %d) = %f\n",id ,i, get(x, i));
							break;
					}
					/**/
				}
				break;
			}
		}
	}

	template<int dir, int var>
	__device__ void update_segment( FTYPE *x, Segment3D &seg, TimeLayer3D_GPU &layer, int id, int num_seg, int max_n_max_n )
	{
		int i = seg.posx;
		int j = seg.posy;
		int k = seg.posz;
		
		if( dir == Z_as_Y ) 
		{
			k = seg.posy;
			j = seg.posz;
		}

		for (int t = 0; t < seg.size; t++)
		{
			switch (var)
			{
			case type_U: layer.elem(layer.u, i, j, k) = get(x,t); break;
			case type_V: layer.elem(layer.v, i, j, k) = get(x,t); break;
			case type_W: layer.elem(layer.w, i, j, k) = get(x,t); break;
			case type_T: layer.elem(layer.T, i, j, k) = get(x,t); break;
			}

			switch (dir)
			{
			case X: i++; break;
			case Y: j++; break;
			case Z: k++; break;
			case Z_as_Y: j++; break;
			}
		}
	}

	template<int dir, int var, int swipe>
	__global__ void solve_segments( FluidParamsGPU p, int num_seg, Segment3D *segs, Node* nodes, TimeLayer3D_GPU cur, TimeLayer3D_GPU temp, TimeLayer3D_GPU next,
									FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int  max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];

		int n = seg.size;

		int start, end;
		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
			start = dimX - n;
			end = dimX - 1;
			break;
		default:
			start = 0;
			end = n-1;
			break;
		}

		switch (swipe)
		{
		case ALL:
		case FORWARD:
			apply_bc0<dir, var>(seg.posx, seg.posy, seg.posz, cur.dimy, cur.dimz, get(b,start), get(c,start), get(d,start), nodes, seg.type);
			apply_bc1<dir, var>(seg.endx, seg.endy, seg.endz, cur.dimy, cur.dimz, get(a,end), get(b,end), get(d,end), nodes, seg.type);
		
			build_matrix<dir, var>(p, seg.posx, seg.posy, seg.posz, a, b, c, d, n, cur, temp, id, num_seg, max_n_max_n, dimX, seg.type);

		case BACK:			
			solve_tridiagonal<dir, swipe>(a, b, c, d, x, n, id, num_seg, max_n_max_n, dimX, seg.type);
			break;
		}
			
		switch (swipe)
		{
		case ALL:
		case BACK:
			update_segment<dir, var>(x, seg, next, id, num_seg, max_n_max_n);
			break;
		}
	}

	template<int dir, int var>
	__global__ void build_matrix( FluidParamsGPU p, int num_seg, Segment3D *segs, Node* nodes, TimeLayer3D_GPU cur, TimeLayer3D_GPU temp, 
								  FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, int max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];

		int n = seg.size;

		int start, end;
		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
			start = dimX - n;
			end = dimX - 1;
			break;
		default:
			start = 0;
			end = n-1;
		}

		apply_bc0<dir, var>(seg.posx, seg.posy, seg.posz, cur.dimy, cur.dimz, get(b,start), get(c,start), get(d,start), nodes, seg.type);
		apply_bc1<dir, var>(seg.endx, seg.endy, seg.endz, cur.dimy, cur.dimz, get(a,end), get(b,end), get(d,end), nodes, seg.type);
		
		build_matrix<dir, var>(p, seg.posx, seg.posy, seg.posz, a, b, c, d, n, cur, temp, id, num_seg, max_n_max_n, dimX);	
	}

	template<int dir, int var, int swipe>
	__global__ void solve_matrix( int num_seg, Segment3D *segs, TimeLayer3D_GPU next,
								  FTYPE *a, FTYPE *b, FTYPE *c, FTYPE *d, FTYPE *x, int max_n_max_n, int dimX, int id_shift = 0 )
	{
		// fetch current segment info
		int id = id_shift + blockIdx.x * blockDim.x + threadIdx.x;
		if( id >= num_seg + id_shift) return;
		Segment3D &seg = segs[id];

		switch (dir)
		{
		case X:
			if (seg.skipX)
				return;
		}

		int n = seg.size;

		solve_tridiagonal<dir, swipe>(a, b, c, d, x, n, id, num_seg, max_n_max_n, seg.type);
		
		switch (swipe)
		{
		case BACK:
			update_segment<dir, var>(x, seg, next, id, num_seg, max_n_max_n);
			break;
		}
	}

	template<DirType dir, VarType var>
	void LaunchSolveSegments_dir_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, Node **nodes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	/*
		Y and Z direction only (and X if nGPUs = 1)
	*/
	{
		GPUplan *pGPUplan = GPUplan::Instance();

		int max_n_max_n;
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );
		dim3 block(SOLVER_BLOCK_DIM);

		for (int i = 0; i < pGPUplan->size(); i++)
		{
			hipSetDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			int dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = pGPUplan->node(i)->getLength1D() * max_n;  // valid for Y and Z direction only if nGPUs > 1

			switch( decomposeOpt )
			{
			case true:
				build_matrix<dir, var><<<grid, block>>>( p, num_seg[i], segs[i], nodes[i], cur, temp, d_a[i], d_b[i], d_c[i], d_d[i], max_n_max_n, dimX );
				break;

			case false:
				//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);
				solve_segments<dir, var, ALL><<<grid, block>>>( p, num_seg[i], segs[i], nodes[i], cur, temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
				break;
			}
		}
		hipDeviceSynchronize();

		if ( decomposeOpt )
		{
			for (int i = 0; i < pGPUplan->size(); i++)
			{
				hipSetDevice(i);
				cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

				int dimX = pGPUplan->node(i)->getLength1D();
				dim3 grid((num_seg[i] + block.x - 1)/block.x);

				max_n_max_n = pGPUplan->node(i)->getLength1D() * max_n;

				solve_matrix<dir, var, ALL><<<grid, block>>>( num_seg[i], segs[i], next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			}
			hipDeviceSynchronize();
		}
		//printf("dir %d: next dd_T after forward and back calculation: %f\n", TestUtil::sumEllementsMultiGPU(next.dd_T, next.dimx * next.dimy * next.dimz, next.haloSize));
		//TestUtil::printEllementsMultiGPU<FTYPE>(next.dd_T, next.dimy, next.dimz, next.dimy*next.dimz, true);
		//fflush(stdout);
	}

	template<VarType var>
	void LaunchSolveSegments_X_var( FluidParamsGPU p, int *num_seg, Segment3D **segs, Node **nodes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
									  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	/*
		X direction only
	*/
	{
		GPUplan *pGPUplan = GPUplan::Instance();
		PARAplan* pplan = PARAplan::Instance();
		int irank =  pplan->rank();
		int size = pplan->size();
		
		if (pGPUplan->size() == 1 && pplan->size() == 1)
		{
			LaunchSolveSegments_dir_var<X, var>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt );
			return;
		}
		/**/

		int max_n_max_n;
		int max_n = max( max( cur.dimx, cur.dimy ), cur.dimz );

		int haloSize = max_n * max_n * MAX_SEGS_PER_ROW;
		int dimX;

		dim3 block(SOLVER_BLOCK_DIM);

		FTYPE *mpi_buf_1 = new FTYPE[haloSize];
		FTYPE *mpi_buf_2 = new FTYPE[haloSize];

		//***************************************
		//cur.SetDevice(0); temp.SetDevice(0); next.SetDevice(0);
		//int dimX = pGPUplan->node(0).getLength1D();
		//dim3 grid((num_seg[0] + block.x - 1)/block.x);

		//max_n_max_n = max_n * max_n;
		//solve_segments<X, var, FORWARD><<<grid, block>>>( p, num_seg[0], segs[0], nodes[0], cur, temp, next, d_a[0], d_b[0], d_c[0], d_d[0], d_x[0], max_n_max_n, dimX );
		////hipDeviceSynchronize();
		//solve_segments<X, var, BACK><<<grid, block>>>( p, num_seg[0], segs[0], nodes[0], cur, temp, next, d_a[0], d_b[0], d_c[0], d_d[0], d_x[0], max_n_max_n, dimX );
		//hipDeviceSynchronize();
		//return;
		//***************************************
		hipSetDevice(0);
		paraDevRecv<FTYPE, FORWARD>(d_c[0], mpi_buf_1, haloSize, 666);
		paraDevRecv<FTYPE, FORWARD>(d_d[0], mpi_buf_2, haloSize, 667);
		for (int i = 0; i < pGPUplan->size(); i++)
		{
			hipSetDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = max_n * max_n;

			//hipFuncSetCacheConfig(reinterpret_cast<const void*>(solve_segments<dir), var, ALL>, hipFuncCachePreferL1);
			solve_segments<X, var, FORWARD><<<grid, block>>>( p, num_seg[i], segs[i], nodes[i], cur, temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i < pGPUplan->size() - 1) //  send to node n+1
			{
				haloMemcpyPeer<FTYPE, FORWARD>( d_c, i, haloSize, dimX * haloSize );
				haloMemcpyPeer<FTYPE, FORWARD>( d_d, i, haloSize, dimX * haloSize );
			}
		}
		//hipDeviceSynchronize();
		hipSetDevice(pGPUplan->size()-1);
		paraDevSend<FTYPE, FORWARD>(d_c[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf_1, haloSize, 666);
		paraDevSend<FTYPE, FORWARD>(d_d[pGPUplan->size()-1] + haloSize + dimX * haloSize - haloSize, mpi_buf_2, haloSize, 667);

		paraDevRecv<FTYPE, BACK>(d_x[pGPUplan->size()-1] + haloSize +  dimX * haloSize, mpi_buf_1, haloSize, 668);
		for (int i = pGPUplan->size() - 1; i >= 0; i--)
		{
			hipSetDevice(i);
			cur.SetDevice(i); temp.SetDevice(i); next.SetDevice(i);

			dimX = pGPUplan->node(i)->getLength1D();
			dim3 grid((num_seg[i] + block.x - 1)/block.x);

			max_n_max_n = max_n * max_n;
			solve_segments<X, var, BACK><<<grid, block>>>( p, num_seg[i], segs[i], nodes[i], cur, temp, next, d_a[i], d_b[i], d_c[i], d_d[i], d_x[i], max_n_max_n, dimX );
			if (i > 0)
				haloMemcpyPeer<FTYPE, BACK>(d_x, i, haloSize, pGPUplan->node(i-1)->getLength1D()*haloSize);
		}
		hipDeviceSynchronize();
		hipSetDevice(0);
		paraDevSend<FTYPE, BACK>(d_x[0] + haloSize, mpi_buf_1, haloSize, 668);		

		delete [] mpi_buf_1;
		delete [] mpi_buf_2;
	}

	template<DirType dir>
	void LaunchSolveSegments_dir( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, Node **nodes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_dir_var<dir, type_U>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_V: LaunchSolveSegments_dir_var<dir, type_V>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_W: LaunchSolveSegments_dir_var<dir, type_W>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_T: LaunchSolveSegments_dir_var<dir, type_T>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		}
	}

	void LaunchSolveSegments_X( FluidParamsGPU p, int *num_seg, Segment3D **segs, VarType var, Node **nodes, TimeLayer3D_GPU &cur, TimeLayer3D_GPU &temp, TimeLayer3D_GPU &next,
								  FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	/*
		Will change later to solve variables in parallel on mGPU (more memory)
	*/
	{
		switch( var )
		{
		case type_U: LaunchSolveSegments_X_var<type_U>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_V: LaunchSolveSegments_X_var<type_V>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_W: LaunchSolveSegments_X_var<type_W>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case type_T: LaunchSolveSegments_X_var<type_T>( p, num_seg, segs, nodes, cur, temp, next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		}
	}

	void SolveSegments_GPU( FTYPE dt, FluidParams params, int *num_seg, Segment3D **segs, VarType var, DirType dir, Node **nodes, TimeLayer3D *cur, TimeLayer3D *temp, TimeLayer3D *next,
							FTYPE **d_a, FTYPE **d_b, FTYPE **d_c, FTYPE **d_d, FTYPE **d_x, bool decomposeOpt )
	{
		TimeLayer3D_GPU d_cur( cur );
		TimeLayer3D_GPU d_temp( temp );
		TimeLayer3D_GPU d_next( next );

		FluidParamsGPU p( var, dir, dt, cur->dx, cur->dy, cur->dz, params );

		switch( dir )
		{
		case X: LaunchSolveSegments_X( p, num_seg, segs, var, nodes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case Y: LaunchSolveSegments_dir<Y>( p, num_seg, segs, var, nodes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case Z: LaunchSolveSegments_dir<Z>( p, num_seg, segs, var, nodes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		case Z_as_Y: LaunchSolveSegments_dir<Z_as_Y>( p, num_seg, segs, var, nodes, d_cur, d_temp, d_next, d_a, d_b, d_c, d_d, d_x, decomposeOpt ); break;
		}
	}
}
