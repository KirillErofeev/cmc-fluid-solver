#include "hip/hip_runtime.h"
/*
 *  Copyright 2010-2011 Nikolai Sakharnykh
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

// GPU implementation of TimeLayer3D

#include "Grid3D.h"

#define COPY_BLOCK_DIM_X		32
#define COPY_BLOCK_DIM_Y		8

#define TRANSPOSE_SMEM_TILE_DIM			16
#define TRANSPOSE_SMEM_BLOCK_ROWS		16

#define TRANSPOSE_CACHE_TILE_DIM		32
#define TRANSPOSE_CACHE_BLOCK_ROWS		8

using namespace FluidSolver3D;

__global__ void copy_grid(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = nodes[id].v.x;
			v[id] = nodes[id].v.y;
			w[id] = nodes[id].v.z;
			T[id] = nodes[id].T;
		}
	}
}

__global__ void clear(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
		{
			u[id] = const_u;
			v[id] = const_v;
			w[id] = const_w;
			T[id] = const_T;
		}
	}
}

__global__ void copy(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = src[id];
	}
}

__global__ void merge(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	int k = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if( k >= dimz || j >= dimy ) return;

	for (int i = 0; i < dimx; i++)
	{
		int id = i * dimy * dimz + j * dimz + k;
		if( nodes[id].type == target )
			dest[id] = ( dest[id] + src[id] ) / 2;
	}
}

__global__ void transpose_shared(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest)
{
	__shared__ FTYPE tile[TRANSPOSE_SMEM_TILE_DIM][TRANSPOSE_SMEM_TILE_DIM+1];

    // read the tile from global memory into shared memory
	int k0 = blockIdx.x * TRANSPOSE_SMEM_TILE_DIM + threadIdx.x;
	int j0 = blockIdx.y * TRANSPOSE_SMEM_TILE_DIM + threadIdx.y;

	int j1 = blockIdx.y * TRANSPOSE_SMEM_TILE_DIM + threadIdx.x;
    int k1 = blockIdx.x * TRANSPOSE_SMEM_TILE_DIM + threadIdx.y;

	int base_u0 = k0 + j0 * dimz;	
    int base_u1 = j1 + k1 * dimy;
	
	for (int i = 0; i < dimx; i++)
	{	
		// read tile from global to shared memory
		for (int row = 0; row < TRANSPOSE_SMEM_TILE_DIM; row += TRANSPOSE_SMEM_BLOCK_ROWS)
			tile[threadIdx.y + row][threadIdx.x] = src[base_u0 + row * dimz];
		base_u0 += dimz * dimy;
	
		__syncthreads();
	
		// write the transposed tile to global memory 
		for (int row = 0; row < TRANSPOSE_SMEM_TILE_DIM; row += TRANSPOSE_SMEM_BLOCK_ROWS)
			dest[base_u1 + row * dimy] = tile[threadIdx.x][threadIdx.y + row];
		base_u1 += dimz * dimy;
	}
}

__global__ void transpose_cache(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest)
{
	int i = blockIdx.y / ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM);
	int blkY = blockIdx.y % ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM);

	int j1 = blkY * TRANSPOSE_CACHE_TILE_DIM + threadIdx.x;
    int k1 = blockIdx.x * TRANSPOSE_CACHE_TILE_DIM + threadIdx.y;

	int base_src = blockIdx.x * TRANSPOSE_CACHE_TILE_DIM + blkY * TRANSPOSE_CACHE_TILE_DIM * dimz;
	int base_dst = j1 + k1 * dimy;
	
	base_src += dimz * dimy * i;
	base_dst += dimz * dimy * i;

	// load directly from global memory filling up L1
	for (int row = 0; row < TRANSPOSE_CACHE_TILE_DIM; row += TRANSPOSE_CACHE_BLOCK_ROWS)
		dest[base_dst + row * dimy] = src[base_src + threadIdx.x * dimz + threadIdx.y + row];
}

void CopyFromGrid_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	copy_grid<<<grid, block>>>(dimx, dimy, dimz, u, v, w, T, nodes, target);
	hipDeviceSynchronize();
}

void CopyFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	copy<<<grid, block>>>(dimx, dimy, dimz, src, dest, nodes, target);
	hipDeviceSynchronize();
}

void MergeFieldTo_GPU(int dimx, int dimy, int dimz, FTYPE *src, FTYPE *dest, Node *nodes, NodeType target)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	merge<<<grid, block>>>(dimx, dimy, dimz, src, dest, nodes, target);
	hipDeviceSynchronize();
}

void Clear_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *v, FTYPE *w, FTYPE *T, Node *nodes, NodeType target, FTYPE const_u, FTYPE const_v, FTYPE const_w, FTYPE const_T)
{
	dim3 block(COPY_BLOCK_DIM_X, COPY_BLOCK_DIM_Y);
	dim3 grid((dimz + block.x - 1)/block.x, (dimy + block.y - 1)/block.y);
	clear<<<grid, block>>>(dimx, dimy, dimz, u, v, w, T, nodes, target, const_u, const_v, const_w, const_T);
	hipDeviceSynchronize();
}

void Transpose_GPU_shared(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *dest_u)
{
	dim3 block(TRANSPOSE_SMEM_TILE_DIM, TRANSPOSE_SMEM_BLOCK_ROWS);
	dim3 grid((dimz + TRANSPOSE_SMEM_TILE_DIM - 1)/TRANSPOSE_SMEM_TILE_DIM, (dimy + TRANSPOSE_SMEM_TILE_DIM - 1)/TRANSPOSE_SMEM_TILE_DIM);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(transpose_shared), hipFuncCachePreferL1);
	transpose_shared<<<grid, block>>>(dimx, dimy, dimz, u, dest_u);
	hipDeviceSynchronize();
}

void Transpose_GPU_cache(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *dest_u)
{
	dim3 block(TRANSPOSE_CACHE_TILE_DIM, TRANSPOSE_CACHE_BLOCK_ROWS);
	dim3 grid((dimz + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM, dimx * ((dimy + TRANSPOSE_CACHE_TILE_DIM - 1)/TRANSPOSE_CACHE_TILE_DIM));
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(transpose_cache), hipFuncCachePreferL1);
	transpose_cache<<<grid, block>>>(dimx, dimy, dimz, u, dest_u);
	hipDeviceSynchronize();
}

void Transpose_GPU(int dimx, int dimy, int dimz, FTYPE *u, FTYPE *dest_u)
{
#if 1
	Transpose_GPU_shared(dimx, dimy, dimz, u, dest_u);
#else
	Transpose_GPU_cache(dimx, dimy, dimz, u, dest_u);
#endif
}